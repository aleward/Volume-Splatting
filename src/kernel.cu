#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"
//#include ""


#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 64 // 32 // 

// TODO Parameters for comparing neighboring splats - check if applies.
// These worked well in 565 reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// Buffers to hold all the splat information.
// These get allocated in Splats::initSimulation.
glm::vec3 *dev_pos;
// Ping-pong buffer example. Unneeded as of now
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// TODO - to use these buffers for neighbor search
// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// Additional buffers for sorting data
glm::vec3 *dev_positionThrust;
glm::vec3 *dev_velocityThrust;

// Grid parameters based on simulation parameters.
// These are automatically computed in Splats::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/*******************
* Important Getters*
*******************/

float Splats::width() {
	return scene_width;
}

float Splats::length() {
	return scene_length;
}

float Splats::height() {
	return scene_height;
}

float Splats::spacing() {
	return scene_spacing;
}

float Splats::scale() {
	return scene_scale;
}

void resetValueCheck(std::string valName, float orig, float curr, float spacing) {
	if (orig != curr) {
		std::cout << "Value for " << valName << " has changed from " << 
			orig << " to " << curr << " due to the chosen scene_spacing of " << 
			spacing << std::endl;
	}
}

void Splats::setDimensions(float w, float l, float h, float s) {
	// TODO Play with memory to get the max particle num higher:
	if (h > 102.0 && s < 1.5) {
		scene_spacing = 1.5f;
		std::cout << "Not enough VRAM for the detail of this scene. " << 
			"Scene_spacing increased to " << scene_spacing << std::endl;
	}
	else if (h <= 102.0 && s == 1.5) {
		scene_spacing = 1.f;
		std::cout << "We can get finer detail at this level so we decreased scene_spacing to " <<
			scene_spacing << std::endl;
	}
	else {
		scene_spacing = s;
	}

	scene_width  = floor(w / scene_spacing) * scene_spacing;
	scene_length = floor(l / scene_spacing) * scene_spacing;
	scene_height = floor(h / scene_spacing) * scene_spacing;

	if (h > 0.0 && scene_height < 1.0) { scene_height = scene_spacing; }

	scene_scale = std::max(std::max(scene_width, scene_length), scene_height);

	// Let the user know that some values may have changed due  to the chosen scene_spacing
	resetValueCheck("scene_width", w, scene_width, scene_spacing);
	resetValueCheck("scene_length", l, scene_length, scene_spacing);
	resetValueCheck("scene_height", h, scene_height, scene_spacing);
}

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

// Splat - setting grid based on DICOM image dimensions
__global__ void kernGenerateInitPosArray(int time, int N, glm::vec3 * arr, float length, float width, float height, float size) { //float scale, float height, float size) { 
	int index = (blockIdx.x * blockDim.x) + threadIdx.x; // maybe make this correspond with images? - nah
	if (index < N) {
		arr[index].z = ((float)(index % (int)height) - height / 2.0f) * size;
		arr[index].y = ((float)((index / (int)height) % (int)length) - length / 2.0f) * size;
		arr[index].x = ((float)(index / ((int)height * (int)length)) - width / 2.0f) * size; 

		// TODO strat - maybe - check if segmented (maybe rewrite the images with alpha channel, or just roll wit black for now)
		// If choosing a threshold for irrelevant pixels, dont change anything with the position, then use stream compaction to ignore it for future kernels?
	}
}
// Replace with a read @ index thing

/**
* Initialize memory, update some globals
*/
void Splats::initSimulation(int N) {
	numObjects = N;
	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

	// Don't forget to hipFree in  Splats::endSimulation.
	hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

	//hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
	//checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");
	//
	//hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
	//checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

	// TODO DEPTH SORTING only if camera has moved since last time
	kernGenerateInitPosArray << <fullBlocksPerGrid, blockSize >> > (1, numObjects,
		dev_pos, scene_length / scene_spacing, scene_width / scene_spacing, 
		scene_height / scene_spacing, scene_spacing * 2.0f);
	checkCUDAErrorWithLine("kernGenerateInitPosArray failed!");

	// TODO - check computing grid params
	//gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
	//int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
	//gridSideCount = 2 * halfSideCount;
	//
	//gridCellCount = gridSideCount * gridSideCount * gridSideCount;
	//gridInverseCellWidth = 1.0f / gridCellWidth;
	//float halfGridWidth = gridCellWidth * halfSideCount;
	//gridMinimum.x -= halfGridWidth;
	//gridMinimum.y -= halfGridWidth;
	//gridMinimum.z -= halfGridWidth;

	// TODO - Allocate index-related buffers here.
	///hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
	///checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

	// TODO - and the space-management needed
	//hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
	//checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");
	//
	//hipMalloc((void**)&dev_gridCellStartIndices, N * sizeof(int));
	//checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");
	//
	//hipMalloc((void**)&dev_gridCellEndIndices, N * sizeof(int));
	//checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");
	//
	//hipMalloc((void**)&dev_positionThrust, N * sizeof(glm::vec3));
	//checkCUDAErrorWithLine("hipMalloc dev_positionThrust failed!");
	//
	//hipMalloc((void**)&dev_velocityThrust, N * sizeof(glm::vec3));
	//checkCUDAErrorWithLine("hipMalloc dev_velocityThrust failed!");

	hipDeviceSynchronize();
}


/******************
* copySplatsToVBO *
******************/

/**
* Copy the splat positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	float c_scale = -1.0f / s_scale;

	if (index < N) {
		vbo[4 * index + 0] = pos[index].x * c_scale;
		vbo[4 * index + 1] = pos[index].y * c_scale;
		vbo[4 * index + 2] = pos[index].z * c_scale;
		vbo[4 * index + 3] = 1.0f;
	}
}

/**
* Wrapper for call to the kernCopysplatsToVBO CUDA kernel.
*/

void Splats::copySplatsToVBO(float *vbodptr_positions) { // TODO add more vbos as needed
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos, vbodptr_positions, scene_scale);
	checkCUDAErrorWithLine("copySplatsToVBO failed!");

	hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* TODO Use this to update splat values
* For each of the `N` splats, update its position based on its current velocity.
*/
// TODO check if should pass by value or reference
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel, float scale) {
	// Update position by velocity
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}
	glm::vec3 thisPos = pos[index];
	thisPos += vel[index] * dt;

	// Wrap the splats around so we don't lose them - unneeded now TODO delete
	//thisPos.x = thisPos.x < -scale ? scale : thisPos.x;
	//thisPos.y = thisPos.y < -scale ? scale : thisPos.y;
	//thisPos.z = thisPos.z < -scale ? scale : thisPos.z;
	//
	//thisPos.x = thisPos.x > scale ? -scale : thisPos.x;
	//thisPos.y = thisPos.y > scale ? -scale : thisPos.y;
	//thisPos.z = thisPos.z > scale ? -scale : thisPos.z;

	pos[index] = thisPos;
}


__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
	return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
	glm::vec3 gridMin, float inverseCellWidth,
	glm::vec3 *pos, int *indices, int *gridIndices) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < N) {
		// - Label each splat with the index of its grid cell.
		glm::vec3 posInGrid = glm::floor((pos[index] - gridMin) * inverseCellWidth);
		gridIndices[index] = gridIndex3Dto1D(posInGrid.x, posInGrid.y, posInGrid.z, gridResolution);

		// - Set up a parallel array of integer indices as pointers to the actual
		//   splat data in pos and vel1/vel2
		indices[index] = index;
	}
}

/*__global__ void kernComputeCoherentIndices(int N, glm::vec3 *positionThrust, 
	glm::vec3 *velocityThrust, glm::vec3 *pos, glm::vec3 *vel, int *indices) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < N) {
		// - Find the index of the current splat pre sorting.
		int thisIdx = indices[index];

		// - Update the related positions and velocities
		positionThrust[index] = pos[thisIdx];
		velocityThrust[index] = vel[thisIdx];
	}
}*/

// Indicates that a cell does not enclose any splats?
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		intBuffer[index] = value;
	}
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
	int *gridCellStartIndices, int *gridCellEndIndices) {
	// TODO check - Identify the start point of each cell in the gridIndices array.
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		int thisIdx = particleGridIndices[index];

		// Check previous index to see if this is a start
		if (thisIdx != particleGridIndices[index - 1]) {
			gridCellStartIndices[thisIdx] = index;
		}

		// Check next index to see if this is an end
		if (thisIdx != particleGridIndices[index + 1]) {
			gridCellEndIndices[thisIdx] = index;
		}
	}
	// This is basically a parallel unrolling of a loop that goes
	// "this index doesn't match the one before it, must be a new cell!"
}

__global__ void kernUpdateVelNeighborSearchScattered(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int *gridCellStartIndices, int *gridCellEndIndices,
	int *particleArrayIndices,
	glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
	// Update a splat's velocity using the uniform grid to reduce
	// the number of splats that need to be checked.

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		// - Identify the grid cell that this particle is in
		glm::vec3 posInGrid = glm::floor((pos[index] - gridMin) * inverseCellWidth);
		int gridCell = gridIndex3Dto1D(posInGrid.x, posInGrid.y, posInGrid.z, gridResolution);

		// - Access each splat in each cell and compute velocity change from
		//   the splats rules, if this splat is within the neighborhood distance.
		glm::vec3 perceivedCenter = glm::vec3();
		float numOfNeigh1 = 0.f;

		glm::vec3 c = glm::vec3();

		glm::vec3 perceivedVel = glm::vec3();
		float numOfNeigh3 = 0.f;

		// - Identify which cells may contain neighbors. This isn't always 8.
		for (int x = imax((int) posInGrid.x - 1, 0); x <= imin((int)posInGrid.x + 1, gridResolution); x++) {
			for (int y = imax((int)posInGrid.y - 1, 0); y <= imin((int)posInGrid.y + 1, gridResolution); y++) {
				for (int z = imax((int)posInGrid.z - 1, 0); z <= imin((int)posInGrid.z + 1, gridResolution); z++) {

					int currCell = gridIndex3Dto1D(x, y, z, gridResolution);

					// - For each cell, read the start/end indices in the splat pointer array.
					int startIdx = gridCellStartIndices[currCell];
					int endIdx = gridCellEndIndices[currCell];

					for (int b = startIdx; b < endIdx; b++) {
						int splatIdx = particleArrayIndices[b];

						if (splatIdx != index) {
							float d = glm::distance(pos[index], pos[splatIdx]);
							// Rule 1: splats fly towards their local perceived center of mass, which excludes themselves
							if (d < rule1Distance) {
								perceivedCenter += pos[splatIdx];
								numOfNeigh1++;
							}

							// Rule 2: splats try to stay a distance d away from each other
							if (d < rule2Distance) {
								c -= (pos[splatIdx] - pos[index]);
							}

							// Rule 3: splats try to match the speed of surrounding splats
							if (d < rule3Distance) {
								perceivedVel += vel1[splatIdx];
								numOfNeigh3++;
							}
						}
					}
				}
			}
		}

		if (numOfNeigh1 > 0) {
			perceivedCenter /= numOfNeigh1;
			perceivedCenter = (perceivedCenter - pos[index]) * rule1Scale;
		}

		c *= rule2Scale;

		if (numOfNeigh3 > 0) {
			perceivedVel /= numOfNeigh3;
			perceivedVel *= rule3Scale;
		}

		// Compute a new velocity based on pos and vel1
		glm::vec3 velChange = vel1[index] + perceivedCenter + c + perceivedVel;

		// - Clamp the speed change before putting the new speed in vel2
		float speed = glm::length(velChange);
		if (speed > maxSpeed) {
			velChange /= speed;
		}

		vel2[index] = velChange;
	}
}

/*__global__ void kernUpdateVelNeighborSearchCoherent(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int *gridCellStartIndices, int *gridCellEndIndices,
	glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
	// This is very similar to kernUpdateVelNeighborSearchScattered,
	// except with one less level of indirection.
	// This should expect gridCellStartIndices and gridCellEndIndices to refer
	// directly to pos and vel1.

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		// - Identify the grid cell that this particle is in
		glm::vec3 thisPos = pos[index];
		glm::vec3 posInGrid = glm::floor((thisPos - gridMin) * inverseCellWidth);
		int gridCell = gridIndex3Dto1D(posInGrid.x, posInGrid.y, posInGrid.z, gridResolution);

		// - Access each splat in each cell and compute velocity change from
		//   the splats rules, if this splat is within the neighborhood distance.
		glm::vec3 perceivedCenter = glm::vec3(0.f);
		float numOfNeigh1 = 0.f;

		glm::vec3 c = glm::vec3();

		glm::vec3 perceivedVel = glm::vec3(0.f);
		float numOfNeigh3 = 0.f;

		// - Identify which cells may contain neighbors. This isn't always 8.
		for (int x = imax((int)posInGrid.x - 1, 0); x <= imin((int)posInGrid.x + 1, gridResolution); x++) {
			for (int y = imax((int)posInGrid.y - 1, 0); y <= imin((int)posInGrid.y + 1, gridResolution); y++) {
				for (int z = imax((int)posInGrid.z - 1, 0); z <= imin((int)posInGrid.z + 1, gridResolution); z++) {

					int currCell = gridIndex3Dto1D(x, y, z, gridResolution);

					// - For each cell, read the start/end indices in the splat pointer array.
					//   DIFFERENCE: For best results, consider what order the cells should be
					//   checked in to maximize the memory benefits of reordering the splats data.
					int startIdx = gridCellStartIndices[currCell];
					int endIdx = gridCellEndIndices[currCell];

					for (int b = startIdx; b <= endIdx; b++) {
						if (b != index) {
							float d = glm::distance(pos[b], thisPos);

							// Rule 1: splats fly towards their local perceived center of mass, which excludes themselves
							if (d < rule1Distance) {
								perceivedCenter += pos[b];
								numOfNeigh1++;
							}

							// Rule 2: splats try to stay a distance d away from each other
							if (d < rule2Distance) {
								c -= (pos[b] - thisPos);
							}

							// Rule 3: splats try to match the speed of surrounding splats
							if (d < rule3Distance) {
								perceivedVel += vel1[b];
								numOfNeigh3++;
							}
						}
					}
				}
			}
		}

		if (numOfNeigh1 > 0) {
			perceivedCenter /= numOfNeigh1;
			perceivedCenter = (perceivedCenter - thisPos) * rule1Scale;
		}

		c *= rule2Scale;

		if (numOfNeigh3 > 0) {
			perceivedVel /= numOfNeigh3;
			perceivedVel *= rule3Scale;
		}

		// Compute a new velocity based on pos and vel1
		glm::vec3 velChange = vel1[index] + perceivedCenter + c + perceivedVel;

		// - Clamp the speed change before putting the new speed in vel2
		float speed = glm::length(velChange);
		if (speed > maxSpeed) {
			velChange /= speed;
			velChange *= maxSpeed;
		}

		vel2[index] = velChange;
	}
}*/

// TODO - A test kernel right now, to be replaced
void Splats::simpleUpdate(float dt) {
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt,
		dev_pos, dev_vel1, scene_scale);
	checkCUDAErrorWithLine("kernUpdatePos failed!");
}

void Splats::stepSimulationScatteredGrid(float dt) {
	// Uniform Grid Neighbor search using Thrust sort.

	// In Parallel:
	// - label each particle with its array index as well as its grid index.
	//   Use 2x width grids.
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, 
		gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, 
		dev_particleArrayIndices, dev_particleGridIndices);
	checkCUDAErrorWithLine("kernComputeIndices failed!");

	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	dev_thrust_particleArrayIndices = thrust::device_pointer_cast(dev_particleArrayIndices);
	dev_thrust_particleGridIndices = thrust::device_pointer_cast(dev_particleGridIndices);

	thrust::sort_by_key(dev_thrust_particleGridIndices, 
		dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of splat indices
	kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (gridCellCount, 
		dev_gridCellStartIndices, INT16_MAX);
	checkCUDAErrorWithLine("kernResetIntBuffer failed!");

	kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (gridCellCount,
		dev_gridCellEndIndices, INT16_MAX);
	checkCUDAErrorWithLine("kernResetIntBuffer2 failed!");

	kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, 
		dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
	checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

	// - Perform velocity updates using neighbor search
	/*kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid, blockSize >> > (numObjects,
		gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, 
		dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, 
		dev_pos, dev_vel1, dev_vel2);
	checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");*/

	// - Update positions
	kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt,
		dev_pos, dev_vel1, scene_scale);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	// - Ping-pong buffers as needed
	glm::vec3 *temp_vel = dev_vel1;
	dev_vel1 = dev_vel2;
	dev_vel2 = temp_vel;
}

/*void Splats::stepSimulationCoherentGrid(float dt) {
	// Uniform Grid Neighbor search using Thrust sort on cell-coherent data.

	// In Parallel:
	// - label each particle with its array index as well as its grid index.
	//   Use 2x width grids.
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects,
		gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos,
		dev_particleArrayIndices, dev_particleGridIndices);
	checkCUDAErrorWithLine("kernComputeIndices failed!");

	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	dev_thrust_particleArrayIndices = thrust::device_pointer_cast(dev_particleArrayIndices);
	dev_thrust_particleGridIndices = thrust::device_pointer_cast(dev_particleGridIndices);

	thrust::sort_by_key(dev_thrust_particleGridIndices,
		dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of splat indices
	kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (gridCellCount,
		dev_gridCellStartIndices, INT16_MAX);
	checkCUDAErrorWithLine("kernResetIntBuffer failed!");

	kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (gridCellCount,
		dev_gridCellEndIndices, INT16_MAX);
	checkCUDAErrorWithLine("kernResetIntBuffer2 failed!");

	kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects,
		dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
	checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

	// - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
	//   the particle data in the simulation array.
	//   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
	kernComputeCoherentIndices << <fullBlocksPerGrid, blockSize >> > (numObjects,
		dev_positionThrust, dev_velocityThrust, dev_pos, dev_vel1, dev_particleArrayIndices);
	checkCUDAErrorWithLine("kernComputeCoherentIndices failed!");

	// Ping pong pos
	glm::vec3 *temp_pos = dev_positionThrust;
	dev_positionThrust = dev_pos;
	dev_pos = temp_pos;

	// - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, blockSize >> > (numObjects,
		gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
		dev_gridCellStartIndices, dev_gridCellEndIndices, dev_positionThrust, 
		dev_velocityThrust, dev_vel2);
	checkCUDAErrorWithLine("kernUpdateVelNeighborSearchCoherent failed!");

	// - Update positions
	kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt,
		dev_pos, dev_vel1, scene_scale);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	// - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
	glm::vec3 *temp_vel = dev_vel1;
	dev_vel1 = dev_vel2;
	dev_vel2 = temp_vel;
}*/

void Splats::endSimulation() {
	//hipFree(dev_vel1);
	//hipFree(dev_vel2);
	hipFree(dev_pos);

	// TODO- Free any additional buffers here.
	///hipFree(dev_particleArrayIndices);
	//hipFree(dev_particleGridIndices);
	//hipFree(dev_gridCellStartIndices);
	//hipFree(dev_gridCellEndIndices);
	//
	//hipFree(dev_positionThrust);
	//hipFree(dev_velocityThrust);
}

void Splats::unitTest() {
	// test unstable sort
	int *dev_intKeys;
	int *dev_intValues;
	int N = 10;

	std::unique_ptr<int[]>intKeys{ new int[N] };
	std::unique_ptr<int[]>intValues{ new int[N] };

	intKeys[0] = 0; intValues[0] = 0;
	intKeys[1] = 1; intValues[1] = 1;
	intKeys[2] = 0; intValues[2] = 2;
	intKeys[3] = 3; intValues[3] = 3;
	intKeys[4] = 0; intValues[4] = 4;
	intKeys[5] = 2; intValues[5] = 5;
	intKeys[6] = 2; intValues[6] = 6;
	intKeys[7] = 0; intValues[7] = 7;
	intKeys[8] = 5; intValues[8] = 8;
	intKeys[9] = 6; intValues[9] = 9;

	hipMalloc((void**)&dev_intKeys, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

	hipMalloc((void**)&dev_intValues, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

	std::cout << "before unstable sort: " << std::endl;
	for (int i = 0; i < N; i++) {
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}

	// How to copy data to the GPU
	hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

	// Wrap device vectors in thrust iterators for use with thrust.
	thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
	thrust::device_ptr<int> dev_thrust_values(dev_intValues);
	// Example for using thrust::sort_by_key
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

	// How to copy data back to the CPU side from the GPU
	hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
	hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("memcpy back failed!");

	std::cout << "after unstable sort: " << std::endl;
	for (int i = 0; i < N; i++) {
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}

	// cleanup
	hipFree(dev_intKeys);
	hipFree(dev_intValues);
	checkCUDAErrorWithLine("hipFree failed!");
	return;
}
